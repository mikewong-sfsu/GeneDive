#include "hip/hip_runtime.h"
/****
     File: findRedsGPU.cu
     Date: 5/15/2018
     By: John Lazzarini
     Compile: nvcc findRedsGPU.cu -o frgpu
     Run: ./findRedsGPU

****/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUMPARTICLES 32768
#define NEIGHBORHOOD .05
#define THREADSPERBLOCK 64

void initPos(float *);
float findDistance(float *, int, int);
__device__ float findDistanceGPU(float *, int, int);
void dumpResults(int index[]);

__global__ void findRedsGPU(float *p, int *numI);

int main() {
   hipEvent_t start, stop;
 float time;

 float *pos;
 int *numReds;

//  NUMPARTICLES = atoi(argv[1]);
//  THREADSPERBLOCK = atoi(argv[2]);

 pos = (float *) malloc(NUMPARTICLES * sizeof(int) * 4);
 numReds = (int *) malloc(NUMPARTICLES * sizeof(int));

 initPos(pos);

 // your code to allocate device arrays for pos and numReds go here
 float* gpuPos;
 int* gpuNumReds;

 hipMalloc((void **)&gpuPos,  NUMPARTICLES * sizeof(int) * 4);
 hipMalloc((void **)&gpuNumReds, NUMPARTICLES * sizeof(int));

 hipMemcpy(gpuPos, pos, NUMPARTICLES * sizeof(int) * 4, hipMemcpyHostToDevice);
 hipMemcpy(gpuNumReds, numReds, NUMPARTICLES * sizeof(int), hipMemcpyHostToDevice);


 // create timer events
 hipEventCreate(&start);
 hipEventCreate(&stop);

 hipEventRecord(start, 0);

 /* invoke kernel findRedsGPU here */
 findRedsGPU<<<NUMPARTICLES/THREADSPERBLOCK, THREADSPERBLOCK>>>(gpuPos, gpuNumReds);

 hipDeviceSynchronize();

 // your code to copy results to numReds[] go here
//  hipMemcpy(pos, gpuPos, posSize, hipMemcpyDeviceToHost);
 hipMemcpy(numReds, gpuNumReds, NUMPARTICLES * sizeof(int), hipMemcpyDeviceToHost);


 hipEventRecord(stop, 0);
 hipEventSynchronize(stop);
 hipEventElapsedTime(&time, start, stop);

 printf("Elapsed time = %f\n", time);

 dumpResults(numReds);
}

void initPos(float *p) {
  // your code for initializing pos goes here
  int i;
  int roll;
  for (i=0; i<NUMPARTICLES; i++) {
    p[i*4] = rand() / (float) RAND_MAX;
    p[i*4+1] = rand() / (float) RAND_MAX;
    p[i*4+2] = rand() / (float) RAND_MAX;
    roll = rand() % 3;
    if (roll == 0)
      p[i*4+3] = 0xff0000;
    else if (roll == 1)
      p[i*4+3] = 0x00ff00;
    else
      p[i*4+3] = 0x0000ff;
 }
}

__device__ float findDistanceGPU(float *p, int i, int j) {

  // your code for calculating distance for particle i and j
  float dx, dy, dz;

  dx = p[i*4] - p[j*4];
  dy = p[i*4+1] - p[j*4+1];
  dz = p[i*4+2] - p[j*4+2];

  return(sqrt(dx*dx + dy*dy + dz*dz));
}

__global__ void findRedsGPU(float *p, int *numI) {

  // your code for counting red particles goes here
  int i;
  int particle = blockDim.x * blockIdx.x + threadIdx.x;
  float distance;
  numI[particle] = 0;

  for (i=0; i<NUMPARTICLES; i++) {
      if (i!=particle) {
        /* calculate distance between particles i, j */
        distance = findDistanceGPU(p, i, particle);
        /* if distance < r and color is red, increment count */
        if (distance < NEIGHBORHOOD && p[i*4+3] == 0xff0000) {
          numI[particle]++;
        }
      }
  }
}


void dumpResults(int index[]) {
  int i;
  FILE *fp;

  fp = fopen("./dump.out", "w");

  for (i=0; i<NUMPARTICLES; i++) {
    fprintf(fp, "%d %d\n", i, index[i]);
  }

  fclose(fp);
}